#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#define BS 8
#define N 10
void print(int *A, int n)
{
  for(int i=0; i<n; i++) printf("%d ",A[i]);
}
__global__ void addition( int *A,int *B,int  *C, int n)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
if(i < n) C[i]=A[i]+B[i];
}
int main(void)
{
 int threadsPerBlock, blocksPerGrid, n, *A,*B,*C,*dA,*dB,*dC;
   n = N; threadsPerBlock = BS;
   blocksPerGrid = (n + BS - 1) / BS;
   A = (int *)malloc(n * sizeof(int));
   B = (int *)malloc(n * sizeof(int));
   C = (int *)malloc(n * sizeof(int));
   for (int i = 0; i < n; i++) A[i] = i * 10; 
   for (int i = 0; i < n; i++) B[i] = i * 20; 
hipMalloc((void **)&dA, n * sizeof(int));
hipMalloc((void **)&dB, n * sizeof(int));
hipMalloc((void **)&dC, n * sizeof(int));
hipMemcpy(dA, A, n * sizeof(int),     hipMemcpyHostToDevice);
hipMemcpy(dB, B, n * sizeof(int),     hipMemcpyHostToDevice);
addition<<<blocksPerGrid, threadsPerBlock>>>(dA,dB,dC,n);
hipMemcpy(C, dC, n * sizeof(int), hipMemcpyDeviceToHost);  
 print(C,n);
 hipFree(dA);  free(A);  
 hipFree(dB);  free(B);
 hipFree(dC);  free(C);

 return 0;
}

