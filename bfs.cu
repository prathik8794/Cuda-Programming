#include "hip/hip_runtime.h"
#include<stdio.h>
#define BS 8

struct Node{
int start;
int length;
};
__global__ void print(int i){
		printf(" %d",i);
}
__global__ void bfs_kernel(Node *Va, int *Ea, bool *Fa, bool *Xa, int *Ca,bool *done,int* n1)
{
int id = threadIdx.x + blockIdx.x*blockDim.x;
	if(id>(*n1))*done=false;

	if(Fa[id] == true && Xa[id]==false)
	{
		printf("%d ",id);
	Fa[id]=false;
	Xa[id]=true;
	__syncthreads();
	int s = Va[id].start;
	int e = s+Va[id].length;
	for(int i=s;i<e;i++){
	int nid = Ea[id];
	if(Xa[nid] == false)
	{	
	Ca[nid]=Ca[id]+1;
	Fa[nid]=true;
	*done=false;
	}
	}
	}
}
int main(){
 int threadsPerBlock, blocksPerGrid;
	threadsPerBlock = 8;
	blocksPerGrid = 1;
int e;
int n;
int* nx = (int *)malloc(sizeof(int));
scanf("%d %d",&n,&e);
*nx = n;
int* nc;
	hipMalloc((void**)&nc, sizeof(int));
	hipMemcpy(nc,nx,sizeof(int), hipMemcpyHostToDevice);
Node node[n];
for(int i=0;i<n;i++){
int x,y;
scanf("%d %d",&x,&y);
node[i].start = x;
node[i].length = y;
}

int edge[e];
for(int i=0;i<e;i++){
int x;
scanf("%d ",&x);
edge[i] = x;
}
bool frontier[n] = {false};
bool visited[n] = {false};
int cost[n] = {0};

int source;
scanf("%d",&source);
frontier[source] = true;

Node* Va;
	hipMalloc((void**)&Va, sizeof(Node)*n);
	hipMemcpy(Va, node, sizeof(Node)*n, hipMemcpyHostToDevice);

int* Ea;
	hipMalloc((void**)&Ea, sizeof(int)*e);
	hipMemcpy(Ea, edge, sizeof(Node)*e, hipMemcpyHostToDevice);

bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool)*n);
	hipMemcpy(Fa, frontier, sizeof(bool)*n, hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool)*n);
	hipMemcpy(Xa, visited, sizeof(bool)*n, hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int)*n);
	hipMemcpy(Ca, cost, sizeof(int)*n, hipMemcpyHostToDevice);

bool done;
bool* dd;
printf("\n\n");
hipMalloc((void**)&dd,sizeof(bool));
int c=0;

printf("Order: \n\n");
do{
	c++;
	done = true;
	hipMemcpy(dd,&done,sizeof(bool),hipMemcpyHostToDevice);
	bfs_kernel<<<blocksPerGrid, threadsPerBlock>>>(Va, Ea, Fa, Xa, Ca,dd,nc);
	hipMemcpy(&done, dd , sizeof(bool), hipMemcpyDeviceToHost);

}while(!done);
hipMemcpy(cost, Ca, sizeof(int)*n, hipMemcpyDeviceToHost);
printf("Number of times the kernel is called : %d \n", c);
printf("\nCost: ");
	for (int i = 0; i<n; i++)
		printf( "%d    ", cost[i]);
	printf("\n");
}
